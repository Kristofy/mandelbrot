#include "hip/hip_runtime.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <cmath>
#include <ctype>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

constexpr int WIDTH = 800;
constexpr int HEIGHT = 600;
constexpr int MAX_ITERATIONS = 1000;
constexpr float ZOOM_FACTOR = 0.9;
constexpr int FPS = 30;

// check if string is a integer (does not check for overflow or negative numbers)
bool is_number(const std::string& s)
{
    std::string::const_iterator it = s.begin();
    while (it != s.end() && std::isdigit(*it)) ++it;
    return !s.empty() && it == s.end();
}

struct Color
{
    uint8_t red;
    uint8_t green;
    uint8_t blue;

    __device__ Color(uint8_t r, uint8_t g, uint8_t b) : red(r), green(g), blue(b) {}
};

// get a "random" color based on the interations after which the iteration producing the mandelbrot set becomes unsable
__device__ Color getColor(int iteration)
{
    int red = (iteration % 8) * 32;
    int green = (iteration % 16) * 16;
    int blue = (iteration % 32) * 8;

    return Color(red, green, blue);
}

// kelnel to quickly generate mandelbrot set interations by pixel
__global__ void generateMandelbrot(uint8_t* image, float zoom, float centerX, float centerY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= WIDTH || y >= HEIGHT)
        return;

    // the actuall mandelbrot set interations, accounting for a shift and a zoom level
    float zx = 0.0;
    float zy = 0.0;
    float cx = centerX + (x - WIDTH / 2) / (0.5 * zoom * WIDTH);
    float cy = centerY + (y - HEIGHT / 2) / (0.5 * zoom * HEIGHT);

    int iteration = 0;
    while (zx * zx + zy * zy < 4.0 && iteration < MAX_ITERATIONS)
    {
        float xtemp = zx * zx - zy * zy + cx;
        zy = 2.0 * zx * zy + cy;
        zx = xtemp;
        iteration++;
    }

    Color color = getColor(iteration);
    image[(y * WIDTH + x) * 3 + 0] = color.red;
    image[(y * WIDTH + x) * 3 + 1] = color.green;
    image[(y * WIDTH + x) * 3 + 2] = color.blue;
}

int main(int argc, char* argv[])
{
    int frames = FPS * (argc > 1 && is_number(argv[1]) ? atoi(argv[1]) : 10);
    
    constexpr dim3 block(16, 16);
    constexpr dim3 grid((WIDTH + block.x - 1) / block.x, (HEIGHT + block.y - 1) / block.y);

    uint8_t* deviceImage;
    hipMalloc(&deviceImage, WIDTH * HEIGHT * 3 * sizeof(uint8_t));
    uint8_t* hostImage = new uint8_t[WIDTH * HEIGHT * 3];
    char filename[256];

    float centerX = -1.4002;  // X-coordinate of the center of the zoom
    float centerY = 0.0;   // Y-coordinate of the center of the zoom

    for (int frame = 0; frame < frames; frame++)
    {
        float zoom = std::sqrt(frame + 1) * ZOOM_FACTOR;
        generateMandelbrot<<<grid, block>>>(deviceImage, zoom, centerX, centerY);
        hipDeviceSynchronize();

        hipMemcpy(hostImage, deviceImage, WIDTH * HEIGHT * 3 * sizeof(uint8_t), hipMemcpyDeviceToHost);
        snprintf(filename, sizeof(filename), "./frames/frame%d.png", frame);
        
        // Writing the Image to disk
        stbi_write_png(filename, WIDTH, HEIGHT, 3, hostImage, WIDTH * 3);
    }

    delete[] hostImage;
    hipFree(deviceImage);

    return 0;
}
